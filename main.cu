
#include <hip/hip_runtime.h>
#include <stdio.h>


#define N 1000


class c {
    public:
    float a = 32;


};



__global__
void add(int *a, int *b, c *c) {
    int i = blockIdx.x;
    int j = blockIdx.y;
    if (i<N) {
        b[i] = c->a;
    }
}



int main() {

    int ha[N], hb[N];
    int *da, *db;

    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));

    for (int i = 0; i<N; ++i) {
        ha[i] = i;
    }

    hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);

    c hc;
    c* dc;

    hipMalloc((void **)&dc, sizeof(c));
    hipMemcpy(dc, &hc, sizeof(c), hipMemcpyHostToDevice);


    add<<<N, 1>>>(da, db, dc);

    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i<N; ++i) {
        printf("%d\n", hb[i]);
    }

    hipFree(da);
    hipFree(db);

    return 0;
}
